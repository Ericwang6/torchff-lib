#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/library.h>
#include <ATen/ATen.h>

#include "common/vec3.cuh"
#include "common/pbc.cuh"


template <typename scalar_t> 
__global__ void build_neighbor_list_nsquared_kernel(
    scalar_t* coords,
    scalar_t* box,
    scalar_t* box_inv,
    scalar_t cutoff2,
    int32_t* pairs,
    int32_t* npairs,
    int32_t natoms,
    int32_t max_npairs
)
{
    int32_t index = threadIdx.x + blockIdx.x * blockDim.x;
    if ( index >= natoms * (natoms - 1) / 2 ) {
        return;
    }
    int32_t i = floor((sqrt(index * 8 + 1) + 1) / 2);
    // if (i * (i - 1) > 2 * index) i--;
    int32_t j = index - (i * (i - 1)) / 2;

    scalar_t drvec[3];
    diff_vec3(&coords[i * 3], &coords[j * 3], drvec);
    apply_pbc_triclinic(drvec, box, box_inv, drvec);

    scalar_t dist2 = drvec[0] * drvec[0] + drvec[1] * drvec[1] + drvec[2] * drvec[2];
    if ( dist2 > cutoff2 ) {
        return;
    }
    int32_t i_pair = atomicAdd(npairs, 1);
    pairs[i_pair * 2] = i;
    pairs[i_pair * 2 + 1] = j;
}


at::Tensor build_neighbor_list_nsquared_cuda(
    at::Tensor& coords,
    at::Tensor& box,
    double cutoff,
    int64_t max_npairs
)
{
    at::Tensor box_inv = at::linalg_inv(box);
    int32_t natoms = coords.size(0);

    int32_t max_npairs_;
    if ( max_npairs == -1 ) {
        max_npairs_ = natoms * (natoms - 1) / 2;
    }
    else {
        max_npairs_ = static_cast<int32_t>(max_npairs);
    }

    int32_t *d_npairs;
    hipMalloc(&d_npairs, sizeof(int32_t));
    hipMemset(d_npairs, 0, sizeof(int32_t));

    int block_dim = 128;
    int grid_dim = (natoms * (natoms - 1) / 2 + block_dim - 1) / block_dim;

    at::Tensor pairs = at::empty({max_npairs_, 2}, coords.options().dtype(at::kInt));
    AT_DISPATCH_FLOATING_TYPES(coords.scalar_type(), "build_neighbor_list_nsquared_cuda", ([&] {
        build_neighbor_list_nsquared_kernel<scalar_t><<<grid_dim, block_dim>>>(
            coords.data_ptr<scalar_t>(),
            box.data_ptr<scalar_t>(),
            box_inv.data_ptr<scalar_t>(),
            static_cast<scalar_t>(cutoff * cutoff),
            pairs.data_ptr<int32_t>(),
            d_npairs,
            natoms,
            max_npairs_
        );
    }));

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    // check if the number of pairs exceeds the capacity
    int32_t npairs_found = 0;
    hipMemcpy(&npairs_found, d_npairs, sizeof(int32_t), hipMemcpyDeviceToHost);
    TORCH_CHECK(npairs_found <= max_npairs_, "Too many neighbor pairs found. Maximum is " + std::to_string(max_npairs_), " but found " + std::to_string(npairs_found));
    hipFree(d_npairs);

    return pairs.index({at::indexing::Slice(0, npairs_found), at::indexing::Slice()});
}


TORCH_LIBRARY_IMPL(torchff, CUDA, m) {
    m.impl("build_neighbor_list_nsquared", build_neighbor_list_nsquared_cuda);
}