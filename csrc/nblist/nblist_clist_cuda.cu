#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/library.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>

#include "common/vec3.cuh"
#include "common/pbc.cuh"


template <typename scalar_t> 
__global__ void assign_cell_index_kernel(
    scalar_t* coords,
    scalar_t* box_inv,
    scalar_t fcrx, scalar_t fcry, scalar_t fcrz, // cell size in fractional coords
    int32_t ncx, int32_t ncy, int32_t ncz, // number of cells in one dimension
    int32_t natoms,
    scalar_t* f_coords,
    int32_t* cell_indices,
    int32_t* natoms_per_cell
)
{
    __shared__ scalar_t s_box_inv[9];
    if ( threadIdx.x < 9 ) {
        s_box_inv[threadIdx.x] = box_inv[threadIdx.x];
    }
    __syncthreads();

    int32_t index = threadIdx.x + blockIdx.x * blockDim.x;
    if ( index >= natoms ) {
        return;
    }

    scalar_t crd[3];
    crd[0] = coords[index * 3];
    crd[1] = coords[index * 3 + 1];
    crd[2] = coords[index * 3 + 2];

    // compute fractional coords
    scalar_t fx = dot_vec3(s_box_inv, crd);
    scalar_t fy = dot_vec3(s_box_inv+3, crd);
    scalar_t fz = dot_vec3(s_box_inv+6, crd);

    // shift to [0, 1]
    fx -= floor(fx);
    fy -= floor(fy);
    fz -= floor(fz);

    // compute cell index
    int32_t cx = (int32_t)(fx / fcrx) % ncx;
    int32_t cy = (int32_t)(fy / fcry) % ncy;
    int32_t cz = (int32_t)(fz / fcrz) % ncz;
    int32_t c = (cx * ncy + cy) * ncz + cz;

    f_coords[index*3]   = fx;
    f_coords[index*3+1] = fy;
    f_coords[index*3+2] = fz;

    cell_indices[index] = c;
    atomicAdd(&natoms_per_cell[c+1], 1);
}


__global__ void compute_cell_prefix(int32_t* sorted_cell_indices, int32_t natoms, int32_t* cell_prefix) {
    int32_t index = threadIdx.x + blockIdx.x * blockDim.x;
    if ( index >= natoms || index == 0 ) {
        return;
    }

    int32_t prev_c = sorted_cell_indices[index - 1];
    int32_t c = sorted_cell_indices[index];
    if ( prev_c != c ) {
        cell_prefix[c] = index;
    }

}


template <typename scalar_t>
__global__ void build_neighbor_list_cell_list_kernel(
    scalar_t* f_coords_sorted, // fractional coordinates sorted by cell index
    scalar_t* box,
    scalar_t cutoff2,
    scalar_t fcrx, scalar_t fcry, scalar_t fcrz,
    int32_t ncx, int32_t ncy, int32_t ncz, // number of cells in each dimension
    int32_t ncr, // number of cells to search in each dimension
    int32_t* sorted_atom_indices, // sorted_atom_indices[i] is the original index of i-th position in f_coords_sorted
    int32_t* cell_prefix,
    int32_t natoms,
    int32_t max_npairs,
    int32_t* pairs,
    int32_t* npairs
)
{
    __shared__ scalar_t s_box[9];
    if ( threadIdx.x < 9 ) {
        s_box[threadIdx.x] = box[threadIdx.x];
    }
    __syncthreads();

    int32_t index = threadIdx.x + blockIdx.x * blockDim.x;

    if ( index >= natoms ) {
        return;
    }

    scalar_t fcrd_i[3];
    fcrd_i[0] = f_coords_sorted[index*3];
    fcrd_i[1] = f_coords_sorted[index*3+1];
    fcrd_i[2] = f_coords_sorted[index*3+2];
    int32_t i = sorted_atom_indices[index];

    int32_t cx = (int32_t)(fcrd_i[0] / fcrx) % ncx;
    int32_t cy = (int32_t)(fcrd_i[1] / fcry) % ncy;
    int32_t cz = (int32_t)(fcrd_i[2] / fcrz) % ncz;

    scalar_t fcrd_j[3];
    scalar_t dfcrd[3];
    scalar_t dcrd[3];

    scalar_t tmp[3];
    int32_t nei_cx, nei_cy, nei_cz, nei_c, i_curr_pair;
    // Loop over neighbor cells
    for (int32_t dcx = -ncr; dcx <= ncr; ++dcx) {
        nei_cx = (cx + dcx + ncx) % ncx;
        for (int32_t dcy = -ncr; dcy <= ncr; ++dcy) {
            nei_cy = (cy + dcy + ncy) % ncy;
            for (int32_t dcz = -ncr; dcz <= ncr; ++dcz) {
                nei_cz = (cz + dcz + ncz) % ncz;
                nei_c = (nei_cx * ncy + nei_cy) * ncz + nei_cz;
                for (int32_t j_sort = cell_prefix[nei_c]; j_sort < cell_prefix[nei_c+1]; ++j_sort) {
                    if ( index > j_sort ) {
                        fcrd_j[0] = f_coords_sorted[j_sort*3];
                        fcrd_j[1] = f_coords_sorted[j_sort*3+1];
                        fcrd_j[2] = f_coords_sorted[j_sort*3+2];
                        diff_vec3(fcrd_i, fcrd_j, dfcrd);
                        dfcrd[0] -= round(dfcrd[0]);
                        dfcrd[1] -= round(dfcrd[1]);
                        dfcrd[2] -= round(dfcrd[2]);

                        dcrd[0] = dot_vec3(dfcrd, s_box);
                        dcrd[1] = dot_vec3(dfcrd, s_box+3);
                        dcrd[2] = dot_vec3(dfcrd, s_box+6);

                        if ( (dcrd[0] * dcrd[0] + dcrd[1] * dcrd[1] + dcrd[2] * dcrd[2]) <= cutoff2 ) {
                            i_curr_pair = atomicAdd(npairs, 1) % max_npairs;
                            pairs[i_curr_pair*2] = i;
                            pairs[i_curr_pair*2+1] = sorted_atom_indices[j_sort];
                        }
                    }
                }
            }
        }
    }
}


std::tuple<at::Tensor, at::Tensor> build_neighbor_list_cell_list_cuda(
    const at::Tensor& coords,
    const at::Tensor& box,
    const at::Scalar& cutoff,
    const at::Scalar& max_npairs,
    const at::Scalar& cell_size,
    bool padding
)
{
    at::Tensor box_inv = at::linalg_inv(box);
    int32_t natoms = coords.size(0);

    int32_t max_npairs_ = max_npairs.toInt();
    max_npairs_ = ( max_npairs_ == -1 ) ? natoms * (natoms - 1) / 2 : max_npairs_;

    at::Tensor box_cpu = box.to(at::kCPU);
    at::Tensor box_len = at::linalg_norm(box_cpu, 2, 0);
    at::Tensor f_cell_size = cell_size / box_len;
    at::Tensor nc = at::floor(box_len / cell_size).to(at::kInt);

    int32_t ncx = nc[0].item<int32_t>();
    int32_t ncy = nc[1].item<int32_t>();
    int32_t ncz = nc[2].item<int32_t>();
    int32_t ncr = ceilf(cutoff.toFloat() / cell_size.toFloat());

    TORCH_CHECK(ncx > 2 * ncr, "Box is too small in dimension x");
    TORCH_CHECK(ncy > 2 * ncr, "Box is too small in dimension y");
    TORCH_CHECK(ncz > 2 * ncr, "Box is too small in dimension z");


    int block_dim = 32;
    int grid_dim = (natoms + block_dim - 1) / block_dim;

    at::Tensor pairs = at::empty({max_npairs_, 2}, coords.options().dtype(at::kInt));
    at::Tensor npairs = at::zeros({1}, coords.options().dtype(at::kInt));

    at::Tensor f_coords = at::empty_like(coords);
    at::Tensor cell_indices = at::empty({natoms}, pairs.options());
    at::Tensor natoms_per_cell = at::zeros({ncx*ncy*ncz+1}, pairs.options());

    at::Tensor sorted_cell_indices;
    at::Tensor sorted_atom_indices;

    auto stream = at::cuda::getCurrentCUDAStream();

    // Step 1: Compute fractional coords and assign cell index for each atom
    AT_DISPATCH_FLOATING_TYPES(coords.scalar_type(), "assign_cell_index", ([&] {
        scalar_t* fcr = f_cell_size.data_ptr<scalar_t>();
        scalar_t fcrx = fcr[0];
        scalar_t fcry = fcr[1];
        scalar_t fcrz = fcr[2];
        assign_cell_index_kernel<scalar_t><<<grid_dim, block_dim, 0, stream>>>(
            coords.data_ptr<scalar_t>(),
            box_inv.data_ptr<scalar_t>(),
            fcrx, fcry, fcrz,
            ncx, ncy, ncz,
            natoms,
            f_coords.data_ptr<scalar_t>(),
            cell_indices.data_ptr<int32_t>(),
            natoms_per_cell.data_ptr<int32_t>()
        );
    }));

    // Step 2: Sort atoms according to cell indices
    std::tie(sorted_cell_indices, sorted_atom_indices) = at::sort(cell_indices);
    at::Tensor f_coords_sorted = f_coords.index_select(0, sorted_atom_indices);

    // Step 3: Compute prefix (cumsum of number of atoms in each cell)
    at::Tensor cell_prefix = at::cumsum(natoms_per_cell, 0).to(at::kInt);
    // std::cout << "Cell prefix:" << cell_prefix << std::endl; 
    
    // Step 4: Do neighbor list search
    AT_DISPATCH_FLOATING_TYPES(coords.scalar_type(), "build_neighbor_list", ([&] {
        scalar_t* fcr = f_cell_size.data_ptr<scalar_t>();
        scalar_t fcrx = fcr[0];
        scalar_t fcry = fcr[1];
        scalar_t fcrz = fcr[2];
        scalar_t cutoff2 = static_cast<scalar_t>(cutoff.toDouble() * cutoff.toDouble());
        build_neighbor_list_cell_list_kernel<scalar_t><<<grid_dim, block_dim, 0, stream>>>(
            f_coords_sorted.data_ptr<scalar_t>(),
            box.data_ptr<scalar_t>(),
            cutoff2,
            fcrx, fcry, fcrz,
            ncx, ncy, ncz,
            ncr,
            sorted_atom_indices.to(at::kInt).data_ptr<int32_t>(),
            cell_prefix.data_ptr<int32_t>(),
            natoms,
            max_npairs_,
            pairs.data_ptr<int32_t>(),
            npairs.data_ptr<int32_t>()
        );
    }));
    
    if ( !padding ) {
        hipError_t err = hipGetLastError();
        TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

        // check if the number of pairs exceeds the capacity
        int32_t npairs_found = npairs[0].item<int32_t>();
        TORCH_CHECK(npairs_found <= max_npairs_, "Too many neighbor pairs found. Maximum is " + std::to_string(max_npairs_), " but found " + std::to_string(npairs_found));
        return std::make_tuple(pairs.index({at::indexing::Slice(0, npairs_found), at::indexing::Slice()}), npairs);

    }
    else {
        return std::make_tuple(pairs, npairs);
    }

}


__device__ __forceinline__ void unravel_3d(int32_t c, int32_t nx, int32_t ny, int32_t nz, int32_t& x, int32_t& y, int32_t& z) {
    x = c / (ny * nz);
    y = c / nz - x * ny;
    z = c % nz;
}

__device__ __forceinline__ int32_t warp_dist(int32_t a, int32_t b, int32_t n) {
    int32_t d = abs(a - b);
    return min(d, n - d);
}

__device__ __forceinline__ bool is_interact(int32_t start_cidx_i, int32_t end_cidx_i, int32_t start_cidx_j, int32_t end_cidx_j, int32_t ncx, int32_t ncy, int32_t ncz, int32_t ncr) {
    int32_t cx_i, cy_i, cz_i;
    int32_t cx_j, cy_j, cz_j;
    bool interact = false;

    for (int32_t ci = start_cidx_i; ci <= end_cidx_i; ++ci) {
        if ( interact ) { break; }
        unravel_3d(ci, ncx, ncy, ncz, cx_i, cy_i, cz_i);
        for (int32_t cj = start_cidx_j; cj <= end_cidx_j; ++cj) {
            unravel_3d(cj, ncx, ncy, ncz, cx_j, cy_j, cz_j);
            interact = warp_dist(cx_i, cx_j, ncx) <= ncr && warp_dist(cy_i, cy_j, ncy) <= ncr && warp_dist(cz_i, cz_j, ncz) <= ncr;
            if ( interact ) { break; }
        }
    }
    return interact;
}


template <typename scalar_t>
__global__ void find_interacting_blocks(
    int32_t* sorted_cell_indices,
    int32_t num_blocks,
    int32_t ncx, int32_t ncy, int32_t ncz,
    int32_t ncr,
    int32_t* interacting_blocks,
    int32_t* num_interacting_blocks
)
{
    int32_t index = threadIdx.x + blockIdx.x * blockDim.x;
    if ( index >= num_blocks * (num_blocks + 1) / 2 ) {
        return;
    }
    int32_t i = (int32_t)floor((sqrt(((double)index) * 8 + 1) - 1) / 2);
    // if (i * (i - 1) > 2 * index) i--;
    int32_t j = index - (i * (i + 1)) / 2;

    int32_t n;

    // n = atomicAdd(num_interacting_blocks, 1);
    // interacting_blocks[n * 2] = i;
    // interacting_blocks[n * 2 + 1] = j;

    if ( i == j ) {
        n = atomicAdd(num_interacting_blocks, 1);
        interacting_blocks[n * 2] = i;
        interacting_blocks[n * 2 + 1] = j;
    }
    else {
        int32_t start_cidx_i = sorted_cell_indices[i * 32];
        int32_t start_cidx_j = sorted_cell_indices[j * 32];
        int32_t end_cidx_i = (i == num_blocks - 1) ? sorted_cell_indices[i * 32 + 31]: ncx * ncy * ncz - 1;
        int32_t end_cidx_j = (j == num_blocks - 1) ? sorted_cell_indices[j * 32 + 31]: ncx * ncy * ncz - 1;
        if ( is_interact(start_cidx_i, end_cidx_i, start_cidx_j, end_cidx_j, ncx, ncy, ncz, ncr) ) {
            n = atomicAdd(num_interacting_blocks, 1);
            interacting_blocks[n * 2] = i;
            interacting_blocks[n * 2 + 1] = j;
        }
    }
}


template <typename scalar_t>
__global__ void build_neighbor_list_cell_list_shared_kernel(
    scalar_t* f_coords_sorted, // fractional coordinates sorted by cell index
    scalar_t* box,
    scalar_t cutoff2,
    int32_t* sorted_atom_indices, // sorted_atom_indices[i] is the original index of i-th position in f_coords_sorted
    int32_t* interacting_blocks,
    int32_t natoms,
    int32_t max_npairs,
    int32_t* pairs,
    int32_t* npairs
)
{
    // Load box into shared memory
    __shared__ scalar_t s_box[9];

    if ( threadIdx.x < 9 ) {
        s_box[threadIdx.x] = box[threadIdx.x];
    }
    __syncthreads();

    // Indices of two interacting blocks
    int32_t x = interacting_blocks[blockIdx.x * 2];
    int32_t y = interacting_blocks[blockIdx.x * 2 + 1];
    scalar_t fcrd_i[3] = {0.0, 0.0, 0.0};
    scalar_t fcrd_j[3] = {0.0, 0.0, 0.0};
    scalar_t fcrd_i_shfl[3] = {0.0, 0.0, 0.0};
    scalar_t dfcrd[3];
    scalar_t dcrd[3];

    int32_t i = -1;
    int32_t j = -1;
    int32_t i_shfl = -1;
    int32_t i_curr_pair;

    int32_t index_i = x * 32 + threadIdx.x;
    int32_t index_j = y * 32 + threadIdx.x;
    if ( index_i < natoms ) {
        fcrd_i[0] = f_coords_sorted[index_i*3];
        fcrd_i[1] = f_coords_sorted[index_i*3+1];
        fcrd_i[2] = f_coords_sorted[index_i*3+2];
        i = sorted_atom_indices[index_i];
    }

    // Diagnonal blocks
    if ( x == y ) {
        for (int32_t srcLane = 0; srcLane < 32; ++srcLane) {
            fcrd_i_shfl[0] = __shfl_sync(0xFFFFFFFFu, fcrd_i[0], srcLane);
            fcrd_i_shfl[1] = __shfl_sync(0xFFFFFFFFu, fcrd_i[1], srcLane);
            fcrd_i_shfl[2] = __shfl_sync(0xFFFFFFFFu, fcrd_i[2], srcLane);
            i_shfl = __shfl_sync(0xFFFFFFFFu, i, srcLane);

            if ( i != -1 && i_shfl != -1 && i > i_shfl ) {
                // printf("Thread %d Check between %d and %d from lane %d\n", threadIdx.x, i, j, srcLane);
                // diff
                diff_vec3(fcrd_i, fcrd_i_shfl, dfcrd);
                // apply pbc
                dfcrd[0] -= round(dfcrd[0]);
                dfcrd[1] -= round(dfcrd[1]);
                dfcrd[2] -= round(dfcrd[2]);
                // compute
                dcrd[0] = dot_vec3(dfcrd, s_box);
                dcrd[1] = dot_vec3(dfcrd, s_box+3);
                dcrd[2] = dot_vec3(dfcrd, s_box+6);
        
                if ( (dcrd[0] * dcrd[0] + dcrd[1] * dcrd[1] + dcrd[2] * dcrd[2]) <= cutoff2 ) {
                    i_curr_pair = atomicAdd(npairs, 1) % max_npairs;
                    pairs[i_curr_pair*2] = i;
                    pairs[i_curr_pair*2+1] = i_shfl;
                }
            }
        }
    }
    else {
        if ( index_j < natoms ) {
            j = sorted_atom_indices[index_j];
            fcrd_j[0] = f_coords_sorted[index_j*3];
            fcrd_j[1] = f_coords_sorted[index_j*3+1];
            fcrd_j[2] = f_coords_sorted[index_j*3+2];
        }

        for (int32_t srcLane = 0; srcLane < 32; ++srcLane) {
            fcrd_i_shfl[0] = __shfl_sync(0xFFFFFFFFu, fcrd_i[0], srcLane);
            fcrd_i_shfl[1] = __shfl_sync(0xFFFFFFFFu, fcrd_i[1], srcLane);
            fcrd_i_shfl[2] = __shfl_sync(0xFFFFFFFFu, fcrd_i[2], srcLane);
            i_shfl = __shfl_sync(0xFFFFFFFFu, i, srcLane);
            if ( i_shfl != -1 && j != -1 ) {
                // diff
                diff_vec3(fcrd_j, fcrd_i_shfl, dfcrd);
                // apply pbc
                dfcrd[0] -= round(dfcrd[0]);
                dfcrd[1] -= round(dfcrd[1]);
                dfcrd[2] -= round(dfcrd[2]);
                // compute
                dcrd[0] = dot_vec3(dfcrd, s_box);
                dcrd[1] = dot_vec3(dfcrd, s_box+3);
                dcrd[2] = dot_vec3(dfcrd, s_box+6);
        
                if ( (dcrd[0] * dcrd[0] + dcrd[1] * dcrd[1] + dcrd[2] * dcrd[2]) <= cutoff2 ) {
                    i_curr_pair = atomicAdd(npairs, 1) % max_npairs;
                    pairs[i_curr_pair*2] = i_shfl;
                    pairs[i_curr_pair*2+1] = j;
                }
            }
        }
    }
}


std::tuple<at::Tensor, at::Tensor> build_neighbor_list_cell_list_shared_cuda(
    const at::Tensor& coords,
    const at::Tensor& box,
    const at::Scalar& cutoff,
    const at::Scalar& max_npairs,
    const at::Scalar& cell_size,
    bool padding
)
{

#ifdef DEBUG
    std::chrono::duration<double> diff;
    auto start_time_prep = std::chrono::steady_clock::now();
#endif

    at::Tensor box_inv = at::linalg_inv(box);
    int32_t natoms = coords.size(0);

    int32_t max_npairs_ = max_npairs.toInt();
    max_npairs_ = ( max_npairs_ == -1 ) ? natoms * (natoms - 1) / 2 : max_npairs_;

    at::Tensor box_cpu = box.to(at::kCPU);
    at::Tensor box_len = at::linalg_norm(box_cpu, 2, 0);
    at::Tensor f_cell_size = cell_size / box_len;
    at::Tensor nc = at::floor(box_len / cell_size).to(at::kInt);

    int32_t ncx = nc[0].item<int32_t>();
    int32_t ncy = nc[1].item<int32_t>();
    int32_t ncz = nc[2].item<int32_t>();
    int32_t ncr = (int32_t)ceilf(cutoff.toFloat() / cell_size.toFloat());

    TORCH_CHECK(ncx > 2 * ncr, "Box is too small in dimension x");
    TORCH_CHECK(ncy > 2 * ncr, "Box is too small in dimension y");
    TORCH_CHECK(ncz > 2 * ncr, "Box is too small in dimension z");

    at::Tensor pairs = at::empty({max_npairs_, 2}, coords.options().dtype(at::kInt));
    at::Tensor npairs = at::zeros({1}, coords.options().dtype(at::kInt));

    at::Tensor f_coords = at::empty_like(coords);
    at::Tensor cell_indices = at::empty({natoms}, pairs.options());
    at::Tensor natoms_per_cell = at::zeros({ncx*ncy*ncz+1}, pairs.options());

    at::Tensor sorted_cell_indices;
    at::Tensor sorted_atom_indices;

    int32_t num_blocks = (natoms + 31) / 32;
    at::Tensor num_interacting_blocks = at::zeros({1}, coords.options().dtype(at::kInt));
    at::Tensor interacting_blocks = at::empty({num_blocks * (num_blocks + 1) / 2, 2}, coords.options().dtype(at::kInt));

#ifdef DEBUG
    auto end_time_prep = std::chrono::steady_clock::now();
    diff = end_time_prep - start_time_prep;
    std::cout << "Prep time: " << diff.count() * 1000 << " ms" << std::endl;
#endif

    auto stream = at::cuda::getCurrentCUDAStream();

    // Step 1: Compute fractional coords and assign cell index for each atom
    int block_dim = 128;
    int grid_dim = (natoms + block_dim - 1) / block_dim;
    AT_DISPATCH_FLOATING_TYPES(coords.scalar_type(), "assign_cell_index", ([&] {
        scalar_t* fcr = f_cell_size.data_ptr<scalar_t>();
        scalar_t fcrx = fcr[0];
        scalar_t fcry = fcr[1];
        scalar_t fcrz = fcr[2];
        assign_cell_index_kernel<scalar_t><<<grid_dim, block_dim, 0, stream>>>(
            coords.data_ptr<scalar_t>(),
            box_inv.data_ptr<scalar_t>(),
            fcrx, fcry, fcrz,
            ncx, ncy, ncz,
            natoms,
            f_coords.data_ptr<scalar_t>(),
            cell_indices.data_ptr<int32_t>(),
            natoms_per_cell.data_ptr<int32_t>()
        );
    }));

    // Step 2: Sort atoms according to cell indices
    std::tie(sorted_cell_indices, sorted_atom_indices) = at::sort(cell_indices);
    at::Tensor f_coords_sorted = f_coords.index_select(0, sorted_atom_indices);

    // Step 3: Find interacting blocks, each block contains 32 atoms
    grid_dim = (num_blocks * (num_blocks + 1) / 2 + block_dim - 1) / block_dim;
    AT_DISPATCH_FLOATING_TYPES(coords.scalar_type(), "find_interacting_blocks", ([&] {
        find_interacting_blocks<scalar_t><<<grid_dim, block_dim, 0, stream>>>(
            sorted_cell_indices.data_ptr<int32_t>(),
            num_blocks,
            ncx, ncy, ncz,
            ncr,
            interacting_blocks.data_ptr<int32_t>(),
            num_interacting_blocks.data_ptr<int32_t>()
        );
    }));
    
    // Step 4: Process interacting blocks to build neighbor list
    block_dim = 32;
    grid_dim = num_interacting_blocks.to(at::kCPU).item().toInt();
    AT_DISPATCH_FLOATING_TYPES(coords.scalar_type(), "build_neighbor_list", ([&] {
        scalar_t cutoff2 = static_cast<scalar_t>(cutoff.toDouble() * cutoff.toDouble());
        build_neighbor_list_cell_list_shared_kernel<scalar_t><<<grid_dim, block_dim, 0, stream>>>(
            f_coords_sorted.data_ptr<scalar_t>(),
            box.data_ptr<scalar_t>(),
            cutoff2,
            sorted_atom_indices.to(at::kInt).data_ptr<int32_t>(),
            interacting_blocks.data_ptr<int32_t>(),
            natoms,
            max_npairs_,
            pairs.data_ptr<int32_t>(),
            npairs.data_ptr<int32_t>()
        );
    }));
    
    if ( !padding ) {
        hipError_t err = hipGetLastError();
        TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

        // check if the number of pairs exceeds the capacity
        int32_t npairs_found = npairs[0].item<int32_t>();
        TORCH_CHECK(npairs_found <= max_npairs_, "Too many neighbor pairs found. Maximum is " + std::to_string(max_npairs_), " but found " + std::to_string(npairs_found));
        return std::make_tuple(pairs.index({at::indexing::Slice(0, npairs_found), at::indexing::Slice()}), npairs);

    }
    else {
        return std::make_tuple(pairs, npairs);
    }

}


TORCH_LIBRARY_IMPL(torchff, CUDA, m) {
    m.impl("build_neighbor_list_cell_list", build_neighbor_list_cell_list_cuda);
    m.impl("build_neighbor_list_cell_list_shared", build_neighbor_list_cell_list_shared_cuda);
}