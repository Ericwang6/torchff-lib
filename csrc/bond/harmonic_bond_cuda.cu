#include "hip/hip_runtime.h"
#include <torch/library.h>
#include <ATen/ATen.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


template <typename scalar_t>
__global__ void harmonic_bond_energy_cuda_kernel(scalar_t* coords, int64_t* pairs, scalar_t* b0, scalar_t* k, scalar_t* ene, int64_t nbonds) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= nbonds) {
        return;
    }
    int offset = index * 2;
    scalar_t* coords_0 = coords + pairs[offset] * 3;
    scalar_t* coords_1 = coords + (pairs[offset + 1]) * 3;
    scalar_t dx = coords_1[0] - coords_0[0];
    scalar_t dy = coords_1[1] - coords_0[1];
    scalar_t dz = coords_1[2] - coords_0[2];
    scalar_t b = sqrt(dx * dx + dy * dy + dz * dz);
    // atomicAdd(ene, pow(b - b0[index] , 2) * k[index] / 2);
    ene[index] = pow(b - b0[index] , 2) * k[index] / 2;
}


template <typename scalar_t>
__global__ void harmonic_bond_energy_grad_cuda_kernel(
    scalar_t* coords, int64_t* pairs, 
    scalar_t* b0, scalar_t* k, 
    scalar_t* coord_grad, scalar_t* b0_grad, scalar_t* k_grad, 
    int64_t nbonds
) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= nbonds) {
        return;
    }
    int offset = index * 2;
    int64_t offset_0 = pairs[offset] * 3;
    int64_t offset_1 = pairs[offset + 1] * 3;
    scalar_t* coords_0 = coords + offset_0;
    scalar_t* coords_1 = coords + offset_1;
    scalar_t dx = coords_1[0] - coords_0[0];
    scalar_t dy = coords_1[1] - coords_0[1];
    scalar_t dz = coords_1[2] - coords_0[2];
    scalar_t b = sqrt(dx * dx + dy * dy + dz * dz);
    
    scalar_t k_ = k[index];
    scalar_t db = (b - b0[index]);
    scalar_t prefix = k_ * db / b; 
    scalar_t gx = dx * prefix;
    scalar_t gy = dy * prefix;
    scalar_t gz = dz * prefix;

    atomicAdd(&coord_grad[offset_0],      -gx);
    atomicAdd(&coord_grad[offset_0 + 1],  -gy);
    atomicAdd(&coord_grad[offset_0 + 2],  -gz);

    atomicAdd(&coord_grad[offset_1],      gx);
    atomicAdd(&coord_grad[offset_1 + 1],  gy);
    atomicAdd(&coord_grad[offset_1 + 2],  gz);

    k_grad[index] = db * db / 2;
    b0_grad[index] = -k_ * db;
}



at::Tensor compute_harmonic_bond_energy_cuda(
    at::Tensor& coords,
    at::Tensor& pairs,
    at::Tensor& b0,
    at::Tensor& k
) {
    int64_t nbonds = pairs.size(0);

    int block_dim = 1024;
    int grid_dim = (nbonds + block_dim - 1) / block_dim;

    auto ene = at::zeros({nbonds}, coords.options());

    AT_DISPATCH_FLOATING_TYPES(coords.scalar_type(), "compute_harmonic_bond_energy_cuda", ([&] {
        harmonic_bond_energy_cuda_kernel<scalar_t><<<grid_dim, block_dim>>>(
            coords.data_ptr<scalar_t>(),
            pairs.data_ptr<int64_t>(),
            b0.data_ptr<scalar_t>(),
            k.data_ptr<scalar_t>(),
            ene.data_ptr<scalar_t>(),
            nbonds
        );
    }));

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));
    hipDeviceSynchronize();

    return at::sum(ene);
}


std::tuple<at::Tensor, at::Tensor, at::Tensor> compute_harmonic_bond_energy_grad_cuda(
    at::Tensor& coords,
    at::Tensor& pairs,
    at::Tensor& b0,
    at::Tensor& k
) {
    int64_t nbonds = pairs.size(0);

    int block_dim = 1024;
    int grid_dim = (nbonds + block_dim - 1) / block_dim;

    auto coord_grad = at::zeros_like(coords, coords.options());
    auto b0_grad = at::zeros_like(b0, b0.options());
    auto k_grad = at::zeros_like(k, k.options());

    AT_DISPATCH_FLOATING_TYPES(coords.scalar_type(), "compute_harmonic_bond_energy_grad_cuda", ([&] {
        harmonic_bond_energy_grad_cuda_kernel<scalar_t><<<grid_dim, block_dim>>>(
            coords.data_ptr<scalar_t>(),
            pairs.data_ptr<int64_t>(),
            b0.data_ptr<scalar_t>(),
            k.data_ptr<scalar_t>(),
            coord_grad.data_ptr<scalar_t>(),
            b0_grad.data_ptr<scalar_t>(),
            k_grad.data_ptr<scalar_t>(),
            nbonds
        );
    }));

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));
    hipDeviceSynchronize();
    return std::make_tuple(coord_grad, b0_grad, k_grad);
}

TORCH_LIBRARY_IMPL(torchff, AutogradCUDA, m) {
    m.impl("compute_harmonic_bond_energy", compute_harmonic_bond_energy_cuda);
    m.impl("compute_harmonic_bond_energy_grad", compute_harmonic_bond_energy_grad_cuda);
}