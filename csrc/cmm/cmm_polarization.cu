#include "hip/hip_runtime.h"
#include <torch/autograd.h>
#include <torch/library.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "common/vec3.cuh"
#include "common/pbc.cuh"
#include "damps.cuh"
#include "ewald/damps.cuh"




template <typename scalar_t>
__device__ __forceinline__ void pairwise_electric_data_multipole_kernel_rank_1(
    scalar_t c0_i,
    scalar_t dx_i, scalar_t dy_i, scalar_t dz_i,
    scalar_t c0_j,
    scalar_t dx_j, scalar_t dy_j, scalar_t dz_j,
    scalar_t drx, scalar_t dry, scalar_t drz,
    scalar_t damp1, scalar_t damp3, scalar_t damp5,
    scalar_t* c0_i_g,
    scalar_t* dx_i_g, scalar_t* dy_i_g, scalar_t* dz_i_g,
    scalar_t* c0_j_g,
    scalar_t* dx_j_g, scalar_t* dy_j_g, scalar_t* dz_j_g
) 
{
    // dr = rj - ri;
    scalar_t drinv = rsqrt_(drx*drx+dry*dry+drz*drz);
    scalar_t drinv2 = drinv * drinv;
    scalar_t drinv3 = drinv2 * drinv;
    scalar_t drinv5 = drinv3 * drinv2;

    drinv *= damp1;
    drinv3 *= damp3;
    drinv5 *= damp5;

    scalar_t tx = -drx * drinv3; 
    scalar_t ty = -dry * drinv3;
    scalar_t tz = -drz * drinv3;
    
    scalar_t txx = 3 * drx * drx * drinv5 - drinv3;
    scalar_t txy = 3 * drx * dry * drinv5;
    scalar_t txz = 3 * drx * drz * drinv5;
    scalar_t tyy = 3 * dry * dry * drinv5 - drinv3;
    scalar_t tyz = 3 * dry * drz * drinv5;
    scalar_t tzz = 3 * drz * drz * drinv5 - drinv3;     
    
    // charge gradient - electric potential
    *c0_i_g = drinv * c0_j + tx * dx_j + ty * dy_j + tz * dz_j;
    *c0_j_g = drinv * c0_i - tx * dx_i - ty * dy_i - tz * dz_i;
    
    // dipole gradient - electric field
    *dx_i_g = -c0_j * tx - txx * dx_j - txy * dy_j - txz * dz_j;
    *dy_i_g = -c0_j * ty - txy * dx_j - tyy * dy_j - tyz * dz_j;
    *dz_i_g = -c0_j * tz - txz * dx_j - tyz * dy_j - tzz * dz_j;

    *dx_j_g = c0_i * tx - txx * dx_i - txy * dy_i - txz * dz_i;
    *dy_j_g = c0_i * ty - txy * dx_i - tyy * dy_i - tyz * dz_i;
    *dz_j_g = c0_i * tz - txz * dx_i - tyz * dy_i - tzz * dz_i;
}


template <typename scalar_t>
__global__ void cmm_polarization_real_kernel(
    scalar_t* coords,
    scalar_t* g_box,
    scalar_t* g_box_inv,
    int32_t* pairs,
    scalar_t* b_elec_ij,
    int32_t* pairs_excl,
    scalar_t* charges,
    scalar_t* dipoles,
    scalar_t* epot,
    scalar_t* efield,
    int32_t npairs,
    int32_t npairs_excl,
    scalar_t ewald_alpha,
    scalar_t rcut_sr,
    scalar_t rcut_lr
)
{
    // Box
    __shared__ scalar_t box[9];
    __shared__ scalar_t box_inv[9];
    if (threadIdx.x < 9) {
        box[threadIdx.x] = g_box[threadIdx.x];
        box_inv[threadIdx.x] = g_box_inv[threadIdx.x];
    }
    __syncthreads();
    int32_t start = threadIdx.x + blockDim.x * blockIdx.x;
    for (int32_t index = start; index < npairs; index += gridDim.x * blockDim.x) {
        int32_t i = pairs[index * 2];
        int32_t j = pairs[index * 2 + 1];
        if ( i < 0 || j < 0 ) {
            continue;
        }
        scalar_t rij[3];
        diff_vec3(coords+j*3, coords+i*3, rij);
        apply_pbc_triclinic(rij, box, box_inv, rij);
        scalar_t dr = norm3d_(rij[0], rij[1], rij[2]);
        if ( dr >= rcut_lr ) { continue; }

        scalar_t c0_i = charges[i]; 
        scalar_t dx_i = dipoles[i*3]; scalar_t dy_i = dipoles[i*3+1]; scalar_t dz_i = dipoles[i*3+2];
        scalar_t c0_j = charges[j];
        scalar_t dx_j = dipoles[j*3]; scalar_t dy_j = dipoles[j*3+1]; scalar_t dz_j = dipoles[j*3+2];

        scalar_t damps[3];
        
        ewald_erfc_damps(dr, ewald_alpha, damps);
        if ( dr < rcut_sr ) {
            scalar_t tmp[3];
            polarization_damps(dr, b_elec_ij[index], tmp);
            damps[0] -= tmp[0]; damps[1] -= tmp[1]; damps[2] -= tmp[2];
        }
        
        scalar_t edata_i[4]; scalar_t edata_j[4];
        pairwise_electric_data_multipole_kernel_rank_1(
            c0_i, dx_i, dy_i, dz_i, c0_j, dx_j, dy_j, dz_j, rij[0], rij[1], rij[2],
            damps[0], damps[1], damps[2],
            edata_i, edata_i+1, edata_i+2, edata_i+3,
            edata_j, edata_j+1, edata_j+2, edata_j+3
        );

        atomicAdd(&epot[i], edata_i[0]);
        atomicAdd(&epot[j], edata_j[0]);
        atomicAdd(&efield[i*3], edata_i[1]); atomicAdd(&efield[i*3+1], edata_i[2]); atomicAdd(&efield[i*3+2], edata_i[3]);
        atomicAdd(&efield[j*3], edata_j[1]); atomicAdd(&efield[j*3+1], edata_j[2]); atomicAdd(&efield[j*3+2], edata_j[3]);
    }

    for (int32_t index = start; index < npairs_excl; index += gridDim.x * blockDim.x) {
        int32_t i = pairs_excl[index * 2];
        int32_t j = pairs_excl[index * 2 + 1];

        scalar_t rij[3];
        diff_vec3(coords+j*3, coords+i*3, rij);
        apply_pbc_triclinic(rij, box, box_inv, rij);
        scalar_t dr = norm3d_(rij[0], rij[1], rij[2]);

        scalar_t c0_i = charges[i];
        scalar_t c0_j = charges[j];

        scalar_t dx_i = dipoles[i*3]; scalar_t dy_i = dipoles[i*3+1]; scalar_t dz_i = dipoles[i*3+2];
        scalar_t dx_j = dipoles[j*3]; scalar_t dy_j = dipoles[j*3+1]; scalar_t dz_j = dipoles[j*3+2];

        scalar_t epot_i = scalar_t(0.0); scalar_t epot_j = scalar_t(0.0);
        scalar_t efield_i[3] = {}; scalar_t efield_j[3] = {};

        scalar_t damps[3];
        ewald_erfc_damps(dr, ewald_alpha, damps);

        scalar_t edata_i[4]; scalar_t edata_j[4];
        pairwise_electric_data_multipole_kernel_rank_1(
            c0_i, dx_i, dy_i, dz_i, c0_j, dx_j, dy_j, dz_j, rij[0], rij[1], rij[2],
            damps[0]-scalar_t(1.0), damps[1]-scalar_t(1.0), damps[2]-scalar_t(1.0),
            edata_i, edata_i+1, edata_i+2, edata_i+3,
            edata_j, edata_j+1, edata_j+2, edata_j+3
        );

        atomicAdd(&epot[i], edata_i[0]);
        atomicAdd(&epot[j], edata_j[0]);
        atomicAdd(&efield[i*3], edata_i[1]); atomicAdd(&efield[i*3+1], edata_i[2]); atomicAdd(&efield[i*3+2], edata_i[3]);
        atomicAdd(&efield[j*3], edata_j[1]); atomicAdd(&efield[j*3+1], edata_j[2]); atomicAdd(&efield[j*3+2], edata_j[3]);
    }
}


void compute_cmm_polarization_real_space_cuda(
    at::Tensor& coords,
    at::Tensor& box,
    at::Tensor& pairs,
    at::Tensor& pairs_excl,
    at::Tensor& b_elec_ij,
    at::Tensor& vec_in,
    at::Scalar ewald_alpha,
    at::Scalar rcut_sr,
    at::Scalar rcut_lr,
    at::Tensor& vec_out
)
{
    at::Tensor box_inv, ignore;
    std::tie(box_inv, ignore) = at::linalg_inv_ex(box, false);

    auto props = at::cuda::getCurrentDeviceProperties();
    auto stream = at::cuda::getCurrentCUDAStream();
    int32_t npairs = pairs.size(0);
    int32_t natoms = coords.size(0);
    int32_t block_dim = 256;
    int32_t grid_dim = std::min(props->maxBlocksPerMultiProcessor*props->multiProcessorCount, (npairs+block_dim-1)/block_dim);

    AT_DISPATCH_FLOATING_TYPES(coords.scalar_type(), "cmm_polarization_real_kernel", ([&] {
        scalar_t* charges_ptr = vec_in.data_ptr<scalar_t>();
        scalar_t* dipoles_ptr = charges_ptr + natoms;
        scalar_t* epot_ptr = vec_out.data_ptr<scalar_t>();
        scalar_t* efield_ptr = epot_ptr + natoms;
        cmm_polarization_real_kernel<scalar_t><<<grid_dim, block_dim, 0, stream>>>(
            coords.data_ptr<scalar_t>(),
            box.data_ptr<scalar_t>(),
            box_inv.data_ptr<scalar_t>(),
            pairs.data_ptr<int32_t>(),
            b_elec_ij.data_ptr<scalar_t>(),
            pairs_excl.data_ptr<int32_t>(),
            charges_ptr,
            dipoles_ptr,
            epot_ptr,
            efield_ptr,
            npairs, 
            static_cast<int32_t>(pairs_excl.size(0)),
            static_cast<scalar_t>(ewald_alpha.toDouble()),
            static_cast<scalar_t>(rcut_sr.toDouble()),
            static_cast<scalar_t>(rcut_lr.toDouble())
        );
    }));
}

TORCH_LIBRARY_IMPL(torchff, AutogradCUDA, m) {
    m.impl("compute_cmm_polarization_real_space", compute_cmm_polarization_real_space_cuda);
}